#include "hip/hip_runtime.h"
/**
 * CUDA MD5 cracker
 * Copyright (C) 2015  Konrad Kusnierz <iryont@gmail.com>
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along
 * with this program; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include <stdio.h>
#include <iostream>
#include <time.h>
#include <string.h>
#include <stdlib.h>
#include <stdint.h>
#include <sstream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <>

#define CONST_WORD_LIMIT 10
#define CONST_CHARSET_LIMIT 100

constexpr char CONST_CHARSET[] = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789"; //brute-force alphabet
#define CONST_CHARSET_LENGTH (sizeof(CONST_CHARSET) - 1)

#define CONST_WORD_LENGTH_MIN 1
#define CONST_WORD_LENGTH_MAX 8

#define TOTAL_BLOCKS 16384UL
#define TOTAL_THREADS 512UL
#define HASHES_PER_KERNEL 128UL

#include "assert.cu"
#include "md5.cu"

constexpr char CONST_SOFIA_CHARSET[] = "0123456789ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz"; //sofia alphabet, 62 meaning chars, DO NOT EDIT!

/* Global variables */
uint8_t g_wordLength;

char g_word[CONST_WORD_LIMIT];
char g_charset[CONST_CHARSET_LIMIT];
char g_cracked[CONST_WORD_LIMIT];

__device__ __constant__ char g_deviceCharset[CONST_CHARSET_LIMIT];
__device__ __constant__ char g_deviceSofiaCharset[62];
__device__ char g_deviceCracked[CONST_WORD_LIMIT];


__device__ __host__ bool next(uint8_t* length, char* word, uint32_t increment){
  uint32_t idx = 0;
  uint32_t add = 0;
  
  while(increment > 0 && idx < CONST_WORD_LIMIT){
    if(idx >= *length && increment > 0){
      increment--;
    }
    
    add = increment + word[idx];
    word[idx] = add % CONST_CHARSET_LENGTH;
    increment = add / CONST_CHARSET_LENGTH;
    idx++;
  }
  
  if(idx > *length){
    *length = idx;
  }
  
  if(idx > CONST_WORD_LENGTH_MAX){
    return false;
  }

  return true;
}

__device__ inline uint16_t calcSofia(uint32_t word32) {
    return (g_deviceSofiaCharset[(((word32 & 0xFF000000) >> 24) + ((word32 & 0x00FF0000) >> 16)) % 62] << 8) + (g_deviceSofiaCharset[(((word32 & 0x0000FF00) >> 8) + (word32 & 0x000000FF)) % 62]);
}

__global__ void sofiaCrack(uint8_t wordLength, char* charsetWord, uint32_t sofiaHash_target1part, uint32_t sofiaHash_target2part){
  uint32_t idx = (blockIdx.x * blockDim.x + threadIdx.x) * HASHES_PER_KERNEL;
  
  /* Thread variables */
  char threadCharsetWord[CONST_WORD_LIMIT];
  char threadTextWord[CONST_WORD_LIMIT];
  uint8_t threadWordLength;
  uint32_t threadHash01, threadHash02, threadHash03, threadHash04;
  
  uint32_t sofiaHash1part, sofiaHash2part;
  uint16_t *sofiaHash16;

  /* Copy everything to local memory */
  memcpy(threadCharsetWord, charsetWord, CONST_WORD_LIMIT);
  memcpy(&threadWordLength, &wordLength, sizeof(uint8_t));
  
  /* Increment current word by thread index */
  next(&threadWordLength, threadCharsetWord, idx);
  
  for(uint32_t hash = 0; hash < HASHES_PER_KERNEL; hash++){
    for(uint32_t i = 0; i < threadWordLength; i++){
      threadTextWord[i] = g_deviceCharset[threadCharsetWord[i]];
    }
    
    md5Hash((unsigned char*)threadTextWord, threadWordLength, &threadHash01, &threadHash02, &threadHash03, &threadHash04);

    sofiaHash16 = (uint16_t *)&sofiaHash1part;
    *sofiaHash16 = calcSofia(threadHash01);
    *(sofiaHash16 + 1) = calcSofia(threadHash02);

    sofiaHash16 = (uint16_t *)&sofiaHash2part;
    *sofiaHash16 = calcSofia(threadHash03);
    *(sofiaHash16 + 1) = calcSofia(threadHash04);
    
    if(sofiaHash_target1part == sofiaHash1part && sofiaHash_target2part == sofiaHash2part){ //check
      memcpy(g_deviceCracked, threadTextWord, threadWordLength);
    }
    
    if(!next(&threadWordLength, threadCharsetWord, 1)){
      break;
    }
  }
}

int main(int argc, char* argv[]){
  /* Check arguments */
  if(argc != 2 || strlen(argv[1]) != 8){
    std::cout << argv[0] << " <sofia_hash>" << std::endl;
    return -1;
  }
  
  /* Amount of available devices */
  int devices;
  ERROR_CHECK(hipGetDeviceCount(&devices));
  
  /* Sync type */
  ERROR_CHECK(hipSetDeviceFlags(hipDeviceScheduleSpin));
  
  /* Display amount of devices */
  std::cout << "Notice: " << devices << " device(s) found" << std::endl;
  
  /* Hash stored as u32 integers */
  uint32_t sofiaHash1part, sofiaHash2part;
  
  /* Parse argument */
  memcpy(&sofiaHash1part, argv[1], 4);
  memcpy(&sofiaHash2part, argv[1] + 4, 4);

  /* Fill memory */
  memset(g_word, 0, CONST_WORD_LIMIT);
  memset(g_cracked, 0, CONST_WORD_LIMIT);
  memcpy(g_charset, CONST_CHARSET, CONST_CHARSET_LENGTH);
  
  /* Current word length = minimum word length */
  g_wordLength = CONST_WORD_LENGTH_MIN;
  
  /* Main device */
  hipSetDevice(0);
  
  /* Time */
  hipEvent_t clockBegin;
  hipEvent_t clockLast;
  hipEvent_t clockSprintBegin;
  hipEvent_t clockSprintLast;
  
  hipEventCreate(&clockBegin);
  hipEventCreate(&clockLast);
  hipEventCreate(&clockSprintBegin);
  hipEventCreate(&clockSprintLast);
  hipEventRecord(clockBegin, 0);
  
  float milliseconds = 0;

  /* Current word is different on each device */
  char** words = new char*[devices];
  
  for(int device = 0; device < devices; device++){
    hipSetDevice(device);
    
    /* Copy to each device */
    ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCharset), g_charset, sizeof(uint8_t) * CONST_CHARSET_LIMIT, 0, hipMemcpyHostToDevice));
    ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceSofiaCharset), CONST_SOFIA_CHARSET, sizeof(uint8_t) * 62, 0, hipMemcpyHostToDevice));
    ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCracked), g_cracked, sizeof(uint8_t) * CONST_WORD_LIMIT, 0, hipMemcpyHostToDevice));
    
    /* Allocate on each device */
    ERROR_CHECK(hipMalloc((void**)&words[device], sizeof(uint8_t) * CONST_WORD_LIMIT));
  }
  
  while(true){
    bool result = false;
    bool found = false;
    
    hipEventRecord(clockSprintBegin, 0);
    for(int device = 0; device < devices; device++){
      hipSetDevice(device);
      
      /* Copy current data */
      ERROR_CHECK(hipMemcpy(words[device], g_word, sizeof(uint8_t) * CONST_WORD_LIMIT, hipMemcpyHostToDevice)); 
    
      /* Start kernel */
      sofiaCrack<<<TOTAL_BLOCKS, TOTAL_THREADS>>>(g_wordLength, words[device], sofiaHash1part, sofiaHash2part);
      
      /* Global increment */
      result = next(&g_wordLength, g_word, TOTAL_THREADS * HASHES_PER_KERNEL * TOTAL_BLOCKS);
    }
    hipEventRecord(clockSprintLast, 0);

    /* Display progress */
    char word[CONST_WORD_LIMIT];
    
    for(int i = 0; i < g_wordLength; i++){
      word[i] = g_charset[g_word[i]];
    }

    hipEventSynchronize(clockSprintLast);
    hipEventElapsedTime(&milliseconds, clockSprintBegin, clockSprintLast);
    
    std::cout << "Notice: currently at " << std::string(word, g_wordLength) << " (" << (uint32_t)g_wordLength << "); sprint duration: " << milliseconds << " ms; speed: " << (TOTAL_THREADS * HASHES_PER_KERNEL * TOTAL_BLOCKS) / milliseconds << " hash/ms" << std::endl;
    
    for(int device = 0; device < devices; device++){
      hipSetDevice(device);
      
      /* Synchronize now */
      hipDeviceSynchronize();
      
      /* Copy result */
      ERROR_CHECK(hipMemcpyFromSymbol(g_cracked, HIP_SYMBOL(g_deviceCracked), sizeof(uint8_t) * CONST_WORD_LIMIT, 0, hipMemcpyDeviceToHost)); 
      
      /* Check result */
      if(found = *g_cracked != 0){     
        std::cout << "Notice: cracked |" << g_cracked << '|' << std::endl;
        break;
      }
    }
    
    if(!result || found){
      if(!result && !found){
        std::cout << "Notice: found nothing (host)" << std::endl;
      }
      
      break;
    }
  }
  
  for(int device = 0; device < devices; device++){
    hipSetDevice(device);
    
    /* Free on each device */
    hipFree((void**)words[device]);
  }
  
  /* Free array */
  delete[] words;
  
  /* Main device */
  hipSetDevice(0);
  
  hipEventRecord(clockLast, 0);
  hipEventSynchronize(clockLast);
  hipEventElapsedTime(&milliseconds, clockBegin, clockLast);
  
  std::cout << "Notice: computation time " << milliseconds / 1000 << " sec" << std::endl;
  
  hipEventDestroy(clockBegin);
  hipEventDestroy(clockLast);
  hipEventDestroy(clockSprintBegin);
  hipEventDestroy(clockSprintLast);
}
